#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("architecture: %d%d\n", prop.major, prop.minor);
    return 0;
}
